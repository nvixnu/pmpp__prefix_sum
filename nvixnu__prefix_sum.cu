#include "hip/hip_runtime.h"
#include <math.h>
#include "nvixnu__prefix_sum.h"


__global__
void nvixnu__kogge_stone_scan_by_block_kernel(double *input, double *output, const int length, double *last_sum){
	extern __shared__ double section_sums[];

	int tid = blockIdx.x*blockDim.x + threadIdx.x;


	if(tid < length){
		section_sums[threadIdx.x] = input[tid];
	}

	unsigned int stride;
	for( stride= 1; stride < blockDim.x; stride *= 2){
		__syncthreads();
		if(threadIdx.x >= stride){
			section_sums[threadIdx.x] += section_sums[threadIdx.x - stride];
		}
	}
	output[tid] = section_sums[threadIdx.x];
	if(last_sum != NULL && threadIdx.x == (blockDim.x - 1)){
		last_sum[blockIdx.x] = section_sums[threadIdx.x];
	}
}

__global__
void nvixnu__brent_kung_scan_by_block_kernel(double *input, double *output, const int length, double *last_sum){
	extern __shared__ double section_sums[];

	int tid = blockIdx.x*blockDim.x + threadIdx.x;

	if(tid < length){
		section_sums[threadIdx.x] = input[tid];
	}

	__syncthreads();


	for(unsigned int stride = 1; stride < blockDim.x; stride *= 2){
		__syncthreads();
		int idx = (threadIdx.x + 1) * 2 * stride - 1;
		if(idx < blockDim.x){
			section_sums[idx] += section_sums[idx - stride];
		}
	}

	for(int stride = blockDim.x/4; stride > 0; stride /=2){
		__syncthreads();
		int idx = (threadIdx.x + 1) * 2 *stride - 1;
		if((idx + stride) < blockDim.x){
			section_sums[idx + stride] += section_sums[idx];
		}
	}
	__syncthreads();

	output[tid] = section_sums[threadIdx.x];
	if(last_sum != NULL && threadIdx.x == (blockDim.x - 1)){
		last_sum[blockIdx.x] = section_sums[threadIdx.x];
	}
}

__global__
void nvixnu__kogge_stone_3_phase_scan_by_block_kernel(double *input, double *output, const int length, const int section_length, double *last_sum){
	extern __shared__ double section_sums[];
	int b_dim = blockDim.x;

	// How many phases we should have in order to load the input array to shared memory in a coalesced manner (corner turning)
	int phases_count = ceil(section_length/(double)b_dim);
	// The subsection length is setted to be equals to the phases_count, in order to use all threads in the subsection scan
	int sub_section_max_length = phases_count;


	// Phase 1: Corner turning to load the input data into shared memory
	for(int i = 0; i < phases_count; i++){
		int shared_mem_index = i*b_dim + threadIdx.x;
		int input_index = blockIdx.x*section_length + shared_mem_index;
		//This comparison could be removed if we handle the last phase separately and using the dynamic blockIndex assignment
		if(input_index < length && shared_mem_index < section_length){
			section_sums[shared_mem_index] = input[input_index];
		}
	}

	__syncthreads();

	//Phase 1: Perform the scan on each sub_section
	for(int i = 1; i < sub_section_max_length; i++){
		int index = threadIdx.x*sub_section_max_length + i;
		if(index < section_length){
			section_sums[index] += section_sums[index -1];
		}
	}

	__syncthreads();


	//Phase 2: Performs the Kogge-Stone scan for the last element of each subsection. This step could be performed also by Brent-Kung scan
	for(int stride= 1; stride < section_length; stride *= 2){
		__syncthreads();
		// sub_section_length*threadIdx.x: Indicates the start position of each subsection
		// sub_section_length -1: The last item in a given subsection
		int last_element = sub_section_max_length*threadIdx.x + sub_section_max_length -1;
		if(threadIdx.x >= stride && last_element < section_length){
			section_sums[last_element] += section_sums[last_element - stride*sub_section_max_length];
		}
	}




	__syncthreads();

	//Phase 3: Adding the last element of previous sub_section
	for(int i = 0; i < sub_section_max_length - 1; i++){
		__syncthreads();
		if(threadIdx.x != 0){
			int index = threadIdx.x*sub_section_max_length + i;
			if(index < section_length){
				section_sums[index] += section_sums[threadIdx.x*sub_section_max_length - 1];
			}
		}
	}

	//Save the data on the output array
	for(int i = 0; i < phases_count; i++){
		int output_index = blockIdx.x*section_length + i*b_dim + threadIdx.x;
		if(i*b_dim + threadIdx.x < section_length){
			output[output_index] = section_sums[i*b_dim + threadIdx.x];
		}
	}

	if(last_sum != NULL && threadIdx.x == 0){
		last_sum[blockIdx.x] = section_sums[section_length - 1];
	}


}

